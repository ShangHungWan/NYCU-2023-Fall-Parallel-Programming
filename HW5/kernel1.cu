#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *img, int resX, int resY, int maxIterations)
{
    int thisX = threadIdx.x;
    int thisY = threadIdx.y;

    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int iteration = 0;
    float xtemp;
    while (x * x + y * y < 4 && iteration < maxIterations)
    {
        xtemp = x * x - y * y + x;
        y = 2 * x * y + y;
        x = xtemp;
        iteration++;
    }

    img[thisX + thisY * resX] = iteration;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *DImg, *HImg;
    int size = resX * resY * sizeof(int);
    HImg = (int *)malloc(size);
    memcpy(HImg, img, size);

    hipMalloc(&DImg, size);
    hipMemcpy(DImg, HImg, size, hipMemcpyHostToDevice);

    dim3 dimGrid(resX, resY);
    dim3 dimBlock(1, 1);
    mandelKernel<<<dimGrid, dimBlock>>>(upperX, upperY, stepX, stepY, DImg, resX, resY, maxIterations);

    hipMemcpy(img, DImg, size, hipMemcpyDeviceToHost);
    hipFree(DImg);
    free(HImg);
}
