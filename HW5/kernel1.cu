#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *img, int resX, int resY, int maxIterations)
{
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    float z_re = x, z_im = y;
    int iteration = 0;
    while (z_re * z_re + z_im * z_im <= 4.f && iteration < maxIterations)
    {
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2 * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
        iteration++;
    }

    img[thisX + thisY * resX] = iteration;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *DImg, *HImg;
    int size = resX * resY * sizeof(int);

    hipMalloc(&DImg, size);
    HImg = (int *)malloc(size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((resX + dimBlock.x - 1) / dimBlock.x, (resY + dimBlock.y - 1) / dimBlock.y);
    mandelKernel<<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, DImg, resX, resY, maxIterations);

    hipMemcpy(HImg, DImg, size, hipMemcpyDeviceToHost);
    memcpy(img, HImg, size);

    free(HImg);
    hipFree(DImg);
}
